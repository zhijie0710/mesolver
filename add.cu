
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;
	// for(int i = 0; i < n; i++)
	for(int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main() {
	int N = 1<<20;

	// float *x = new float[N];
	// float *y = new float[N];

	float *x, *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for(int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1)/blockSize;
	add<<<numBlocks, blockSize>>>(N, x, y);

	hipDeviceSynchronize();

	float maxError = 0.0f;
	for(int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// delete[] x;
	// delete[] y;

	hipFree(x);
	hipFree(y);

	return 0;
}